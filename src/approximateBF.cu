#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "timerc.h"

#define THREADPERBLOCK 1024

__global__ void approximate_brute_force(char* text, char* pattern, int* match, int pattern_size, int text_size, int k) {
    int pid = threadIdx.x + blockIdx.x * blockDim.x;

    	if (pid <= text_size - pattern_size){
            int flag = 0; 
            for (int i = 0; i < pattern_size; i++){
                if (text[pid+i] != pattern[i]){
                        flag += 1;
			if(flag>k){
				flag=-1
				break;
			}
                }
            }
            match[pid] = flag;
	}
}

int main() {

    //GET THE WORD
    FILE* file_w = fopen("pattern.txt", "r");
    fseek(file_w, 0L, SEEK_END);
    int pattern_size = ftell(file_w);
    rewind(file_w);
    char* pattern = (char*)malloc(pattern_size * sizeof(char));
    fgets(pattern, pattern_size, file_w);
    fclose(file_w);
    pattern_size = strlen(pattern);

    //GET THE TEXT FILE
    FILE* file_s = fopen("text_string.txt", "r");
    fseek(file_s, 0L, SEEK_END);
    int size = ftell(file_s);
    rewind(file_s);
    char* text = (char*)malloc(size * sizeof(char));
    fgets(text, size, file_s);
    fclose(file_s);
    size = strlen(text);


    //CALCULATIONS FOR BLOCK AND THREAD NUMBERS
    int threadnumber = size - pattern_size + 1;
    int blocknumber = 1;
    if (threadnumber > THREADPERBLOCK) {
        blocknumber = (threadnumber / THREADPERBLOCK);
        if (threadnumber % THREADPERBLOCK != 0) {
            blocknumber++;
        }
        threadnumber = THREADPERBLOCK;
    }

    int k = 0;
    k = (pattern_size * 99)/100;

    //PRINT INFORMATION
    printf("Word to find: <%s> - is a placeholder\n", pattern);
    printf("Pattern length is:  %d, Text length is: %d\n", pattern_size, size);
    printf("\n");
    printf("Thread count: %d, Block count: %d\n", threadnumber, blocknumber);


    /*initialized match array*/
    int* match;
    match = (int*)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) {
        match[i] = -1;
    }

    float gpuTime0;
    float gpuTime;
    float gpuTime1;


    /* GPU init*/
    //text buffer in device
    char* dev_text;
    //pattern buffer in device
    char* dev_pattern;
    // match buffer in device
    int* dev_match;
    //output buffer in device
    int* dev_output;


    gstart();
    hipMalloc((void**)&dev_text, size * sizeof(char));
    hipMalloc((void**)&dev_pattern, pattern_size * sizeof(char));
    hipMalloc((void**)&dev_match, size * sizeof(int));
    //hipMalloc((void **)&dev_output, sizeof(int)*size);

    hipMemcpy(dev_text, text, size * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_pattern, pattern, pattern_size * sizeof(char), hipMemcpyHostToDevice);
    gend(&gpuTime0);

    gstart();

    approximate_brute_force << <number_of_blocks, THREADPERBLOCK >> > (dev_text, dev_pattern, dev_match, pattern_size, size, k);
    gend(&gpuTime);

    gstart();
    hipMemcpy(match, dev_match, size * sizeof(int), hipMemcpyDeviceToHost);

    gend(&gpuTime1);


    /*free memory*/
    hipFree(dev_text);
    hipFree(dev_pattern);
    hipFree(dev_match);
    hipFree(dev_output);

    free(text);
    free(pattern);
    free(match);

    printf("GPUTIME0: %f, GPUTIME: %f, GPUTIME1:%f", gpuTime0, gpuTime, gpuTime1);

}